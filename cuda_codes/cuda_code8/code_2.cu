#include <stdio.h>
#include <hip/hip_runtime.h>
#define FULL_MASK 0xffffffff

__global__ void find_sum(const int start, const int end, const float* arr, float* result){
    const int index = threadIdx.x + start;
    register float value = 0.f;
    const int laneId = threadIdx.x % 32;
    const int n = end - start + 1;
        
    // handle last n%32 elements seperately
    if(threadIdx.x >= n / 32 * 32){
        if(laneId == 0) for(int i = index; i <= end; i++) value += arr[i];
    }else{
        value = arr[index];
        for(int offset = 16; offset >= 1; offset /= 2) value += __shfl_down_sync(FULL_MASK, value, offset);
    }

    __shared__ float blockSum;
    if(threadIdx.x == 0) blockSum = 0.0f;
    __syncthreads();
    if(laneId == 0) atomicAdd(&blockSum, value);
    __syncthreads();
    if(threadIdx.x == 0) *result = blockSum;
}

#define N 1024

__global__ void parent_kernel(const float* arr, float* res){
    const int start = 20;
    const int end = 1023;
    if(blockIdx.x == 0 && threadIdx.x == 0) find_sum <<< 1, end - start + 1 >>> (start, end, arr, res + 1);
}

int main(){
    float* arr = (float*)malloc(N * sizeof(float));
    for(int i = 0; i < N; i++) arr[i] = 1;
    // arr[100] = -1;

    float* d_arr;
    hipMalloc(&d_arr, N * sizeof(float));
    hipMemcpy(d_arr, arr, N * sizeof(float), hipMemcpyHostToDevice);

    float* d_res;
    hipMalloc(&d_res, 2*sizeof(float));
    hipMemset(d_res, 0, 2*sizeof(float));
    
    parent_kernel <<< 10, 10 >>> (d_arr, d_res);
    
    float* res = (float *)malloc(2*sizeof(float));
    hipMemcpy(res, d_res, 2*sizeof(float), hipMemcpyDeviceToHost);

    printf("res = %f, %f\n", res[0], res[1]);

    return 0;
}