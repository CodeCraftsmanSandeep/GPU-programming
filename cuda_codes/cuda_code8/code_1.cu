#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 32

//  thread with laneId 31 will have the final sum value
__global__ void findSum(const int n, int* arr){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    register int value = arr[i];
    int laneId = threadIdx.x & 0xffffffff;
    int completed = 0;
    for(int i = 16; i >= 1; i /= 2){
        completed += i;
        int down_value = __shfl_up_sync(0xffffffff, value, i);
        if(laneId >= completed) value += down_value; 
    }
    printf("threadId = %d, blockId = %d, value = %d\n", threadIdx.x, blockIdx.x, value);
}

__global__ void initialize_arr(const int n, int* arr){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    arr[i] = i;
}

int main(){
    int* d_arr;
    hipMalloc(&d_arr, N*sizeof(int));
    initialize_arr <<< 1, 32 >>> (N, d_arr);
    findSum <<< 1, 32 >>> (N, d_arr);
    hipDeviceSynchronize();

    return 0;
}