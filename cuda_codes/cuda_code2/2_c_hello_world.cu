#include <stdio.h>
#include <hip/hip_runtime.h>

// kernel
__global__ void cuda_kernel(){
    printf("Hello world\n");
}

int main(){
    cuda_kernel <<< 1, 5>>>(); // kernel launch (or) kernel invocation
    hipDeviceSynchronize();   // this is must to make host wait until device completed its tasks
    return 0;
}