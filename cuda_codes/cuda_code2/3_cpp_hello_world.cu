#include <bits/stdc++.h> // these many libraries are imported to just check whether everything is cool!! or not
#include <hip/hip_runtime.h>
using namespace std;

__global__ void cuda_kernel(){
    // cout << threadIdx.x << ") " << "Hello world\n";
    printf("thread %d, block %d) Hello world\n", threadIdx.x, blockIdx.x);  // Use printf instead of cout
}

int main(){
    cuda_kernel <<< 2, 4>>> ();
    cout << "Cpp code\n";
    cout << "\n";
    hipDeviceSynchronize();
}

/*
The output may vary because the scheduling of thread blocks across Streaming Multiprocessors (SMs) is arbitrary.

For example, one possible output might be:
    Cpp code:

    thread 0, block 1) Hello world
    thread 1, block 1) Hello world
    thread 2, block 1) Hello world
    thread 3, block 1) Hello world
    thread 0, block 0) Hello world
    thread 1, block 0) Hello world
    thread 2, block 0) Hello world
    thread 3, block 0) Hello world

While another possible output could be:
    Cpp code:

    thread 0, block 0) Hello world
    thread 1, block 0) Hello world
    thread 2, block 0) Hello world
    thread 3, block 0) Hello world
    thread 0, block 1) Hello world
    thread 1, block 1) Hello world
    thread 2, block 1) Hello world
    thread 3, block 1) Hello world
*/