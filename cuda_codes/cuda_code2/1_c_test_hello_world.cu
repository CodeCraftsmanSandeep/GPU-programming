#include <stdio.h>
#include <hip/hip_runtime.h>

// kernel
__global__ void cuda_kernel(){
    printf("Hello world\n");
}

int main(){
    cuda_kernel <<< 1, 2>>>(); // kernel launch (or) kernel invocation
    return 0;
}


// this code will not give any output
// think why?
// see next code to get answer