#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
using namespace std;

#define MAX_ITER 1000  // Maximum number of iterations
#define DAMPING_FACTOR 0.85
#define THRESHOLD 1e-5

__global__ void pageRankKernel(const int *row_ptr, const int *col_idx, const float *rank, float *new_rank, int num_nodes) {
    int v = blockIdx.x * blockDim.x + threadIdx.x;
    if (v < num_nodes) {
        float sum = 0.0f;
        for (int j = row_ptr[v]; j < row_ptr[v + 1]; j++) {
            int u = col_idx[2*j];                 // u -> v is edge in graph
            int out_degree = col_idx[2*j + 1];
            sum +=  rank[u] / out_degree;
        }
        new_rank[v] = (1.0f - DAMPING_FACTOR) / num_nodes + DAMPING_FACTOR * sum;
    }
}

__global__ void initializePageRank(float* rank, int num_nodes){
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u < num_nodes) rank[u] = 1.0f / num_nodes;
}

void pageRank(const int *row_ptr, const int *col_idx, int num_nodes, int num_edges) {
    int num_blocks = (num_nodes + 255) / 256;
    float *d_rank, *d_new_rank;
    int *d_row_ptr, *d_col_idx;

    hipMalloc(&d_rank, num_nodes * sizeof(float)); 
    initializePageRank <<<num_blocks, 256>>> (d_rank, num_nodes);

    hipMalloc(&d_row_ptr, (num_nodes + 1) * sizeof(int));
    hipMalloc(&d_col_idx, 2*num_edges * sizeof(int));
    hipMalloc(&d_new_rank, num_nodes * sizeof(float));

    hipMemcpyAsync(d_row_ptr, row_ptr, (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_col_idx, col_idx, 2*num_edges * sizeof(int), hipMemcpyHostToDevice);

    bool is_old = true;
    for (int i = 0; i < MAX_ITER; i++) {
        if(is_old) pageRankKernel<<<num_blocks, 256>>>(d_row_ptr, d_col_idx, d_rank, d_new_rank, num_nodes);
        else  pageRankKernel<<<num_blocks, 256>>>(d_row_ptr, d_col_idx, d_new_rank, d_rank, num_nodes);
        is_old = !(is_old);
    }

    float rank[num_nodes];
    if(is_old) hipMemcpy(rank, d_rank, num_nodes * sizeof(float), hipMemcpyDeviceToHost);
    else hipMemcpy(rank, d_new_rank, num_nodes * sizeof(float), hipMemcpyDeviceToHost); 

    printf("Final page rank values:\n");
    for(int u = 0; u < num_nodes; u++) printf("pageRank[%d] = %f\n", u, rank[u]);

    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_rank);
    hipFree(d_new_rank);
}

// assumptions:
// 1) the graph is unweighted, directed.
// 2) the graph may have multiple edges, self loops.

int main() {
    int num_nodes, num_edges;
    scanf("%d %d", &num_nodes, &num_edges);

    vector <vector <int>> in_neighbours(num_nodes);
    int* out_degree = (int*)calloc(num_nodes, sizeof(int));

    for(int edge = 0; edge < num_edges; edge++){
        int u, v;
        scanf("%d %d", &u, &v);
        in_neighbours[v].push_back(u);
        out_degree[u]++;
    }

    int in_neighbour_index[num_nodes + 1];  // Row array in CSR format
    int in_neighbour[2*num_edges];            // Col array in CSR format

    int edge = 0;
    in_neighbour_index[0] = 0;
    for(int v = 0; v < num_nodes; v++){
        for(int& u: in_neighbours[v]){
            in_neighbour[2*edge] = u;
            in_neighbour[2*edge + 1] = out_degree[u];
            edge++;
        }
        in_neighbour_index[v+1] = in_neighbour_index[v] + in_neighbours[v].size();
    }

    // Call PageRank function
    pageRank(in_neighbour_index, in_neighbour, num_nodes, num_edges);

    return 0;
}