#include <iostream>
#include <hip/hip_runtime.h>
#define N 100
using namespace std;

__global__ void my_kernel(){
    printf("%d\n", threadIdx.x);
}

int main(){
    my_kernel <<< 1, N>>> ();
    hipDeviceSynchronize();
    return 0;
}

// 0 - 99 need not be printed in serial
// the order is arbitary