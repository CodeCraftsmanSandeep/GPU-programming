#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

__global__ void find(int* x, int* y, int n){
    int i = blockIdx.x;
    int j = threadIdx.x;

    if(i < j){
        computation[i*blockDimx.x + j] = sqrt(x[i] - x[j])**2 + 
    }
}

int threads = n * (n-1)/2;
int blocksize = 32;
int grids = (threads + blocksize - 1) / blocksize;
find <<< n, n >>> ()
 <<< (n+31/32, n + 31/32), (32, 32) >>>

 int i = blockIdx.x*blockDimx.x + threadIdx.x;
 int j = blockIdx.y*blockDimx.y + threadIdx.y;
 