#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void compute_dis(int *x, int *y, double* dis, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(i < n && j < n){
    if(i < j){
      int dis_x = x[i] - x[j];
      int dis_y = y[i] - y[j];
      dis[i*n + j] = sqrt(dis_x * dis_x + dis_y * dis_y);
      printf("%lf\n", dis[i*n + j]);
    }else dis[i*n+j] = 0;
  }
}

template <typename T>
__global__ void compute_max(T* arr, int n, int chunck_size, T* result){
  // each thread finds the maximum of chunck_size number of elements
  int start = blockIdx.x * blockDim.x * chunck_size + threadIdx.x * chunck_size;

  if(start < n){
    T max_ele = arr[start];
    for(int i = start + 1; i <= start + chunck_size - 1 && i < n; i++){
      if(arr[i] > max_ele) max_ele = arr[i];
    }
    printf("thread: %d\n", blockIdx.x * blockDim.x + threadIdx.x);
    result[blockIdx.x * blockDim.x + threadIdx.x] = max_ele;
  }
}

// finding maximum of a n-sized array
template <typename T>
T find_max(T* d_arr, int n){
  T* swap_arr;
  hipMalloc(&swap_arr, n*sizeof(T));

  // chunck_size: is elements alloted to each cuda thread
  int chunck_size = 16;

  // elements alloted to each thread block is: threads_per_block * chunck_size (here = 512 * 16 = 8192)
  int threads_per_block = 512;
  int total_chunck = chunck_size * threads_per_block;

  bool use_swap_arr = false;
  while(n > 1){
    int num_blocks = (n + total_chunck - 1)/ total_chunck;
    cout << "num_blocks = " << num_blocks << "\n";
    if(use_swap_arr == false) compute_max <<< num_blocks, threads_per_block >>> (d_arr, n, chunck_size, swap_arr);
    else compute_max <<< num_blocks, threads_per_block >>> (swap_arr, n, chunck_size, d_arr);
    use_swap_arr = !use_swap_arr;
    n = (n + chunck_size - 1)/chunck_size;
  }
  hipDeviceSynchronize();

  T* max_ptr = (T*)malloc(sizeof(T));
  if(use_swap_arr == false) hipMemcpy(max_ptr, d_arr, sizeof(T), hipMemcpyDeviceToHost);
  else hipMemcpy(max_ptr, swap_arr, sizeof(T), hipMemcpyDeviceToHost);

  return *max_ptr;
}

int main(){
  int n;
  cin >> n;

  // taking input from user
  int x[n];
  int y[n];
  for(int i = 0; i < n; i++){
    cin >> x[i] >> y[i];
  }

  int *dx;
  hipMalloc(&dx, n * sizeof(int));
  hipMemcpyAsync(dx, x, n * sizeof(int), hipMemcpyHostToDevice);

  int *dy;
  hipMalloc(&dy, n * sizeof(int));
  hipMemcpyAsync(dy, y, n * sizeof(int), hipMemcpyHostToDevice);

  double *dis;
  hipMalloc(&dis, n * n * sizeof(double));

  dim3 block(32, 32);
  dim3 grid((n + 31)/ 32, (n + 31)/ 32);

  compute_dis <<< grid, block >>> (dx, dy, dis, n);
  hipDeviceSynchronize();

  // finding maximum of a n*n sized array
  // find_max has cudaDeviceSynchronize() at the end
  double max_ele = find_max(dis, n*n);

  cout << "Maximum euclidian distance: " << max_ele << "\n";

  hipFree(dx);
  hipFree(dy);

  return 0;
}