#include <iostream>
#include <hip/hip_runtime.h>
#include <iomanip>

template <typename T>
__global__ void compute_max(T* arr, int n, int chunck_size, T* result){
  // each thread finds the maximum of chunck_size number of elements
  int start = blockIdx.x * blockDim.x * chunck_size + threadIdx.x * chunck_size;

  if(start < n){
    T max_ele = arr[start];
    for(int i = start + 1; i <= start + chunck_size - 1 && i < n; i++){
      if(arr[i] > max_ele) max_ele = arr[i];
    }
    // printf("thread: %d\n", blockIdx.x * blockDim.x + threadIdx.x);
    result[blockIdx.x * blockDim.x + threadIdx.x] = max_ele;
  }
}

// finding maximum of a n-sized array
template <typename T>
T find_max(T* arr, int n){
  T* d_arr;
  hipMalloc(&d_arr, n*sizeof(T));
  hipMemcpyAsync(d_arr, arr, n*sizeof(T), hipMemcpyHostToDevice);

  T* swap_arr;
  hipMalloc(&swap_arr, n*sizeof(T));

  // chunck_size: is elements alloted to each cuda thread
  int chunck_size = 16;

  // elements alloted to each thread block is: threads_per_block * chunck_size (here = 512 * 16 = 8192)
  int threads_per_block = 512;
  int total_chunck = chunck_size * threads_per_block;

  bool use_swap_arr = false;
  while(n > 1){
    int num_blocks = (n + total_chunck - 1)/ total_chunck;
    if(use_swap_arr == false) compute_max <<< num_blocks, threads_per_block >>> (d_arr, n, chunck_size, swap_arr);
    else compute_max <<< num_blocks, threads_per_block >>> (swap_arr, n, chunck_size, d_arr);
    use_swap_arr = !use_swap_arr;
    n = (n + chunck_size - 1)/chunck_size;
  }
  hipDeviceSynchronize();

  T* max_ptr = (T*)malloc(sizeof(T));
  if(use_swap_arr == false) hipMemcpy(max_ptr, d_arr, sizeof(T), hipMemcpyDeviceToHost);
  else hipMemcpy(max_ptr, swap_arr, sizeof(T), hipMemcpyDeviceToHost);

  return *max_ptr;
}

int main(){
    int n;
    std::cin >> n;

    double *arr = (double *)malloc(n*sizeof(double));
    for(int i = 0; i < n; i++) std::cin >> std::setprecision(10) >> arr[i];

    std::cout << std::setprecision(10) << find_max(arr, n) << "\n";
}