#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

#define N 1024

__global__ void add_vectors(int* vec1, int* vec2, int* vec){
    vec[threadIdx.x] = vec1[threadIdx.x] + vec2[threadIdx.x];
}

int main(){
    int vec1[N];
    for(int i = 0; i < N; i++) vec1[i] = i; // even this can be paralleized

    int vec2[N];
    for(int i = 0; i < N; i++) vec2[i] = i;

    int vec[N];

    int* vec1_gpu; 
    hipError_t err = hipMalloc(&vec1_gpu, N*sizeof(int));
    if(err != hipSuccess){
        cout << "Cuda memory allocation failed\n";
        return 1;
    }

    // destination, course, size, direction
    hipMemcpyAsync(vec1_gpu, vec1, N*sizeof(int), hipMemcpyHostToDevice);
    // vec1_gpu is pointer is present in stack of cpu, but points to memeory allocated on gpu 

    int* vec2_gpu;
    err = hipMalloc(&vec2_gpu, N*sizeof(int));
    if(err != hipSuccess){
        cout << "Cuda memory allocation failed\n";
        return 1;
    }
    hipMemcpyAsync(vec2_gpu, vec2, N*sizeof(int), hipMemcpyHostToDevice);

    int* vec_gpu;
    err = hipMalloc(&vec_gpu, N*sizeof(int));
    if(err != hipSuccess){
        cout << "Cuda memory allocation failed\n";
        return 1;
    }
    
    add_vectors <<< 1, N>>> (vec1_gpu, vec2_gpu, vec_gpu);
    hipMemcpy(vec, vec_gpu, N*sizeof(int), hipMemcpyDeviceToHost); 
    // cudaMemcpy() is synchronous invocation of cudaMemcpy, so there is no need to use cudaDeviceSynchronize()

    cout << "Vector 1:\n";
    for(int i = 0; i < N; i++) cout << vec1[i] <<  " ";
    cout << "\n\n";

    cout << "Vector 2:\n";
    for(int i = 0; i < N; i++) cout << vec2[i] << " ";
    cout << "\n\n";

    cout << "Added vector:\n";
    for(int i = 0; i < N; i++) cout << vec[i] << " ";
    cout << "\n";

    return 0;
}

// there is also cudaMallocAsync() function
// but for using that we need streams
// we will understand this later